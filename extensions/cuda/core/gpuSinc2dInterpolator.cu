#include "hip/hip_runtime.h"
//
// Author: Liang Yu
// Copyright 2018
//

#include <iostream>
#include <stdio.h>
#include <valarray>
#include <hip/hip_runtime.h>
#include "gpuInterpolator.h"
#include "gpuComplex.h"
#include "../hip/hip_runtime_api.h"

using isce::cuda::core::gpuInterpolator;
using isce::cuda::core::gpuSinc2dInterpolator;
using isce::cuda::core::gpuComplex;


/*
   each derived class needs it's own wrapper_d, gpuInterpolator_g...
*/

template <class U>
__host__ gpuSinc2dInterpolator<U>::gpuSinc2dInterpolator(int sincLen, int sincSub) :
        kernel_length(sincSub), kernel_width(sincLen), sinc_half(sincLen/2), 
        owner(true) {
    // Temporary valarray for storing sinc coefficients
    std::valarray<double> filter(0.0, sincSub * sincLen + 1);
    sinc_coef(1.0, sincLen, sincSub, 0.0, 1, filter);

    // Normalize filter
    for (size_t i = 0; i < sincSub; ++i) {
        // Compute filter sum
        double ssum = 0.0;
        for (size_t j = 0; j < sincLen; ++j) {
            ssum += filter[i + sincSub*j];
        }
        // Normalize the filter
        for (size_t j = 0; j < sincLen; ++j) {
            filter[i + sincSub*j] /= ssum;
        }
    }

    // Copy transpose of filter coefficients to member kernel matrix
    Matrix<double> h_kernel;
    h_kernel.resize(sincSub, sincLen);
    for (size_t i = 0; i < sincLen; ++i) {
        for (size_t j = 0; j < sincSub; ++j) {
            h_kernel(j,i) = filter[j + sincSub*i];
        }
    }

    // Malloc device-side memory (this API is host-side only)
    checkCudaErrors(hipMalloc(&kernel, filter.size()*sizeof(double)));

    // Copy Orbit data to device-side memory and keep device pointer in gpuOrbit object. Device-side 
    // copy constructor simply shallow-copies the device pointers when called
    checkCudaErrors(hipMemcpy(kernel, &(h_kernel[0]), filter.size()*sizeof(double), hipMemcpyHostToDevice));
}


template <class U>
__device__ void wrapper_d(gpuSinc2dInterpolator<U> interp, double x, double y, const U *z, U *value, size_t nx, size_t ny=0) {
    /*
     *  device side wrapper used to get map interfaces of actual device function to global test function
     */
    *value = interp.interpolate(x, y, z, nx, ny); 
}


template <class U>
__global__ void gpuInterpolator_g(gpuSinc2dInterpolator<U> interp, double *x, double *y, const U *z, U *value, size_t nx, size_t ny=0) {
    /*
     *  GPU kernel to test interpolate() on the device for consistency.
     */
    int i = threadIdx.x;
    wrapper_d(interp, x[i], y[i], z, &value[i], nx, ny);
}


template <class U>
__host__ void gpuSinc2dInterpolator<U>::interpolate_h(const Matrix<double>& truth, Matrix<U>& m, double start, double delta, U* h_z) {
    /*
     *  CPU-side function to call the corresponding GPU function on a single thread for consistency checking
        truth = indices to interpolate to
        m = chip
        start, delta = unused
        h_z = output
     */

    // allocate host side memory
    size_t size_input_pts = truth.length() * sizeof(double);
    size_t size_output_pts = truth.length() * sizeof(U);
    double *h_x = (double *)malloc(size_input_pts);
    double *h_y = (double *)malloc(size_input_pts);

    // assign host side inputs
    for (size_t i = 0; i < truth.length(); ++i) {
        h_x[i] = truth(i,0);
        h_y[i] = truth(i,1);
    }

    size_t nx = m.width();
    size_t ny = m.length();

    // allocate device side memory
    double *d_x;
    checkCudaErrors(hipMalloc((void**)&d_x, size_input_pts));
    double *d_y;
    checkCudaErrors(hipMalloc((void**)&d_y, size_input_pts));
    U *d_z;
    checkCudaErrors(hipMalloc((void**)&d_z, size_output_pts));
    U *d_m;
    checkCudaErrors(hipMalloc((U**)&d_m, m.length()*m.width()*sizeof(U)));

    // copy input data
    checkCudaErrors(hipMemcpy(d_x, h_x, size_input_pts, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, h_y, size_input_pts, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m, &m.data()[0], m.length()*m.width()*sizeof(U), hipMemcpyHostToDevice)); 

    // launch!
    int n_threads = truth.length();
    gpuInterpolator_g<U><<<1, n_threads>>>(*this, d_x, d_y, d_m, d_z, nx, ny);
    
    // copy device results to host
    checkCudaErrors(hipMemcpy(h_z, d_z, size_output_pts, hipMemcpyDeviceToHost));

    // free memory
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_z));
    checkCudaErrors(hipFree(d_m));
}


template <class U>
__host__ void 
gpuSinc2dInterpolator<U>::
        sinc_coef(double beta, double relfiltlen, int decfactor, double pedestal, int weight, std::valarray<double> & filter) { 

    int filtercoef = int(filter.size()) - 1;
    double wgthgt = (1.0 - pedestal) / 2.0;
    double soff = (filtercoef - 1.) / 2.;

    double wgt, s, fct;
    for (int i = 0; i < filtercoef; i++) {
        wgt = (1. - wgthgt) + (wgthgt * cos((M_PI * (i - soff)) / soff));
        s = (floor(i - soff) * beta) / (1. * decfactor);
        fct = ((s != 0.) ? (sin(M_PI * s) / (M_PI * s)) : 1.);
        filter[i] = ((weight == 1) ? (fct * wgt) : fct);
    }
}


template <class U>
__device__ U gpuSinc2dInterpolator<U>::interpolate(double x, double y, const U* chip, size_t nx, size_t ny) {
    /*
    definitions with respect to ResampSlc interpolate and sinc_eval_2d
    x   := fracAz
    y   := fracRg
    z   := chip
    nx  := chip length
    ny  := chip width
    */
    // Initialize return value
    U ret(0.0);

    // Separate interpolation coordinates into integer and fractional components
    const int ix = __double2int_rd(x);
    const int iy = __double2int_rd(y);
    const double frpx = x - ix;
    const double frpy = y - iy;

    if (!((ix < sinc_half) || (ix > (ny - sinc_half))) || 
         ((iy < sinc_half) || (iy > (nx - sinc_half)))) {
    
        // Modify integer interpolation coordinates for sinc evaluation
        const int intpx = ix + sinc_half - 1;
        const int intpy = iy + sinc_half - 1;

        // Interpolate for valid indices
        if ((intpx >= (kernel_width-1)) && (intpx < nx) && (intpy >= (kernel_width-1)) && (intpy < ny)) {
            // Get nearest kernel indices
            int ifracx = min(max(0, int(frpx*kernel_length)), kernel_length-1);
            int ifracy = min(max(0, int(frpy*kernel_length)), kernel_length-1);
            // Compute weighted sum
            for (int i = 0; i < kernel_width; i++) {
                for (int j = 0; j < kernel_width; j++) {
                    ret += chip[(intpy-i)*nx + intpx - j]
                         * kernel[ifracy*kernel_width + i]
                         * kernel[ifracx*kernel_width + j];
                }
            }
        }
    }
    // Done
    return ret;
}


template <class U>
__host__ __device__ gpuSinc2dInterpolator<U>::~gpuSinc2dInterpolator() {
#ifndef __CUDA_ARCH__
    if (owner)
        checkCudaErrors(hipFree(kernel));
#endif
}

/*
 each template parameter needs it's own declaration here
 */
template class gpuSinc2dInterpolator<double>;
template class gpuSinc2dInterpolator<gpuComplex<double>>;
template class gpuSinc2dInterpolator<gpuComplex<float>>;

template __global__ void
gpuInterpolator_g<double>(gpuSinc2dInterpolator<double> interp, double *x, double *y,
                                  const double *z, double *value, size_t nx, size_t ny);
template __global__ void
gpuInterpolator_g<gpuComplex<double>>(gpuSinc2dInterpolator<gpuComplex<double>> interp, double *x, double *y,
                                  const gpuComplex<double> *z, gpuComplex<double> *value, size_t nx, size_t ny);
template __global__ void
gpuInterpolator_g<gpuComplex<float>>(gpuSinc2dInterpolator<gpuComplex<float>> interp, double *x, double *y,
                                  const gpuComplex<float> *z, gpuComplex<float> *value, size_t nx, size_t ny);
