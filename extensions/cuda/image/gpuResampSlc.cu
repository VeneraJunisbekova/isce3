#include "hip/hip_runtime.h"
//-*- coding: utf-8 -*-
//
// Author: Liang Yu
// Copyright: 2018
#include <math.h>

// isce::core
#include "isce/core/Constants.h"
#include "isce/core/Interpolator.h"
#include "isce/core/Poly2d.h"

// isce::cuda::core
#include "isce/cuda/core/gpuPoly2d.h"

// isce::cuda::image
#include "gpuResampSlc.h"
#include "gpuImageMode.h"

#include "isce/cuda/hip/hip_runtime_api.h"
#include <fstream>
#include <string>
using isce::cuda::core::gpuComplex;
using isce::cuda::core::gpuPoly2d;
using isce::cuda::core::gpuInterpolator;
using isce::cuda::core::gpuSinc2dInterpolator;
using isce::cuda::image::gpuImageMode;

#define THRD_PER_BLOCK 512// Number of threads per block (should always %32==0)

__global__
void transformTile(const gpuComplex<float> *tile,
                   gpuComplex<float> *chip,
                   gpuComplex<float> *imgOut,
                   const float *rgOffTile,
                   const float *azOffTile,
                   const gpuPoly2d rgCarrier,
                   const gpuPoly2d azCarrier,
                   const gpuPoly2d doppler,
                   gpuImageMode mode,       // image mode for image to be resampled
                   gpuImageMode refMode,    // image mode for reference master image
                   gpuSinc2dInterpolator<gpuComplex<float>> interp,
                   bool flatten,
                   int outWidth,
                   int outLength,
                   int inWidth,
                   int inLength,
                   int chipSize,
                   int rowOffset) {

    int iTileOut = blockDim.x * blockIdx.x + threadIdx.x;
    int iChip = iTileOut * chipSize * chipSize;                                          
    int chipHalf = chipSize/2;

    if (iTileOut < outWidth*outLength) {
        int i = iTileOut / outWidth;
        int j = iTileOut % outWidth;
        imgOut[iTileOut] = gpuComplex<float>(0., 0.);
        //imgOut[iTileOut] = tile[iTileOut];

        // Unpack offsets
        const float azOff = azOffTile[iTileOut];
        const float rgOff = rgOffTile[iTileOut];

        // Break into fractional and integer parts
        const int intAz = __float2int_rd(i + azOff);
        const int intRg = __float2int_rd(j + rgOff);
        const double fracAz = i + azOff - intAz;
        const double fracRg = j + rgOff - intRg;
       
        // Check bounds again. Use rowOffset to account tiles where tile.rowStart != tile.firstRowImage
        bool intAzInBounds = !((intAz+rowOffset < chipHalf) || (intAz >= (inLength - chipHalf)));
        bool intRgInBounds = !((intRg < chipHalf) || (intRg >= (inWidth - chipHalf)));

        int i_dbg = 62250;
        //if (iTileOut % i_dbg == 0)
        //    printf("RiB %d, AiB %d, intAz %d, chipHalf %d, outLength %d\n", 
        //            intRgInBounds, intAzInBounds, intAz, chipHalf, inLength);
        if (intAzInBounds && intRgInBounds) {
        //if (false) {
            // evaluate Doppler polynomial
            const double dop = doppler.eval(0, j) * 2 * M_PI / mode.prf;

            // Doppler to be added back. Simultaneously evaluate carrier that needs to
            // be added back after interpolation
            double phase = (dop * fracAz) 
                + rgCarrier.eval(i + azOff, j + rgOff) 
                + azCarrier.eval(i + azOff, j + rgOff);

            // Flatten the carrier phase if requested
            if (flatten && refMode.isRefMode) {
                phase += ((4. * (M_PI / mode.wavelength)) * 
                    ((mode.startingRange - refMode.startingRange) 
                    + (j * (mode.rangePixelSpacing - refMode.rangePixelSpacing)) 
                    + (rgOff * mode.rangePixelSpacing))) + ((4.0 * M_PI 
                    * (refMode.startingRange + (j * refMode.rangePixelSpacing))) 
                    * ((1.0 / refMode.wavelength) - (1.0 / mode.wavelength)));
            }
            
            // Modulate by 2*PI
            phase = fmod(phase, 2.0*M_PI);
            
            // Read data chip without the carrier phases
            for (int ii = 0; ii < chipSize; ++ii) {
                // Row to read from
                const int chipRow = intAz + ii - chipHalf + rowOffset;
                // Carrier phase
                const double phase = dop * (ii - 4.0);
                const gpuComplex<float> cval(cos(phase), -sin(phase));
                if (iTileOut % i_dbg == 0)
                    printf("i%d j%d cR%d iA%d ii%d cH%d| ", i, j, chipRow, intAz, ii, chipHalf);
                // Set the data values after removing doppler in azimuth
                for (int jj = 0; jj < chipSize; ++jj) {
                    // Column to read from
                    const int chipCol = intRg + jj - chipHalf;
                    chip[iChip + ii*chipSize+jj] = tile[chipRow*outWidth+chipCol] * cval;
                    gpuComplex<float> tile_val = tile[chipRow*outWidth+chipCol];
                    if (iTileOut % i_dbg == 0)
                        printf("%f,%f ", tile_val.r, tile_val.i);
                        //printf("%d ", chipCol);
                }
                if (iTileOut % i_dbg == 0)
                    printf("\n");
            }

            // Interpolate chip
            //const gpuComplex<float> cval(1., 1.);
            const gpuComplex<float> cval = interp.interpolate(
                chipHalf + fracRg + 1, chipHalf + fracAz + 1, &chip[iChip], chipSize, chipSize
            );

            // Add doppler to interpolated value and save
            imgOut[iTileOut] = cval * gpuComplex<float>(cos(phase), sin(phase));
        }
    }
}


// Interpolate tile to perform transformation
void isce::cuda::image::
gpuTransformTile(isce::image::Tile<std::complex<float>> & tile,
               isce::io::Raster & outputSlc,
               isce::image::Tile<float> & rgOffTile,
               isce::image::Tile<float> & azOffTile,
               const isce::core::Poly2d & rgCarrier,
               const isce::core::Poly2d & azCarrier,
               const isce::core::Poly2d & doppler,
               isce::product::ImageMode mode,       // image mode for image to be resampled
               isce::product::ImageMode refMode,    // image mode for reference master image
               bool haveRefMode,
               isce::cuda::core::gpuSinc2dInterpolator<gpuComplex<float>> interp,
               int inWidth, int inLength, bool flatten, int chipSize) {

    // Cache geometry values
    const int outWidth = azOffTile.width();
    const int outLength = azOffTile.length();

    // Allocate valarray for output image block
    std::valarray<std::complex<float>> imgOut(outLength * outWidth);
    // Initialize to zeros
    imgOut = std::complex<float>(0.0, 0.0);

    // declare equivalent objects in device memory
    gpuComplex<float> *d_tile;
    gpuComplex<float> *d_chip;
    gpuComplex<float> *d_imgOut;
    float *d_rgOffTile, *d_azOffTile;
    gpuPoly2d d_rgCarrier(rgCarrier);
    gpuPoly2d d_azCarrier(azCarrier);
    gpuImageMode d_mode(mode);
    gpuImageMode d_refMode;
    if (haveRefMode)
        gpuImageMode d_mode(refMode);
    gpuPoly2d d_doppler(doppler);

    // determine sizes
    size_t nInPixels = (tile.lastImageRow() - tile.firstImageRow() + 1) * outWidth;
    size_t nOutPixels = imgOut.size();
    size_t nOutBytes = nOutPixels * sizeof(gpuComplex<float>);
    size_t nChipBytes = nOutBytes * chipSize * chipSize;

    // allocate equivalent objects in device memory
    checkCudaErrors(hipMalloc(&d_tile, nInPixels*sizeof(gpuComplex<float>)));
    checkCudaErrors(hipMalloc(&d_chip, nChipBytes));
    checkCudaErrors(hipMalloc(&d_imgOut, nOutBytes));
    checkCudaErrors(hipMalloc(&d_azOffTile, nInPixels*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_rgOffTile, nInPixels*sizeof(float)));

    // copy objects to device memory
    checkCudaErrors(hipMemcpy(d_tile, &tile[0], nInPixels*sizeof(gpuComplex<float>), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_azOffTile, &azOffTile[0], nInPixels*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rgOffTile, &rgOffTile[0], nInPixels*sizeof(float), hipMemcpyHostToDevice));

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((nOutPixels+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    printf("rowStart=%d outWidth=%d outLength=%d inLength=%d firstImageRow=%d lastImageRow=%d imgOut\n",
            tile.rowStart(),outWidth,outLength,inLength,
            tile.firstImageRow(),tile.lastImageRow());
    // global call to transform
    transformTile<<<grid, block>>>(d_tile, 
                                   d_chip,
                                   d_imgOut, 
                                   d_rgOffTile, 
                                   d_azOffTile, 
                                   d_rgCarrier, 
                                   d_azCarrier, 
                                   d_doppler, 
                                   d_mode, 
                                   d_refMode,
                                   interp,
                                   flatten,
                                   outWidth,
                                   outLength,
                                   inWidth,
                                   inLength,
                                   chipSize,
                                   tile.rowStart()-tile.firstImageRow());

    // Check for any kernel errors
    checkCudaErrors(hipPeekAtLastError());

    // copy to host memory
    checkCudaErrors(hipMemcpy(&imgOut[0], d_imgOut, nOutBytes, hipMemcpyDeviceToHost));

    if (outLength != 500) {
        std::string fname = "gpu_"+std::to_string(outLength)+"_"+std::to_string(tile.rowStart())+"_.bin";        
        std::ofstream ofile(fname, std::ios::binary);
        ofile.write((char*)&imgOut[0], nOutBytes);
    }
    for (int i = 0; i < 10; ++i)
        printf("%f,%f ", std::real(imgOut[i]), std::imag(imgOut[i]));
    printf("\n");

    // deallocate to device memory
    checkCudaErrors(hipFree(d_tile));
    checkCudaErrors(hipFree(d_chip));
    checkCudaErrors(hipFree(d_imgOut));
    checkCudaErrors(hipFree(d_azOffTile));
    checkCudaErrors(hipFree(d_rgOffTile));
    
    // Write block of data
    outputSlc.setBlock(imgOut, 0, tile.rowStart(), outWidth, outLength);
}
