#include "hip/hip_runtime.h"
//-*- coding: utf-8 -*-
//
// Author: Liang Yu
// Copyright: 2018

// isce::core
#include "isce/core/Constants.h"
#include "isce/core/Interpolator.h"
#include "isce/core/Poly2d.h"

// isce::cuda::core
#include "isce/cuda/core/gpuComplex.h"
#include "isce/cuda/core/gpuInterpolator.h"
#include "isce/cuda/core/gpuPoly2d.h"

// isce::cuda::image
#include "gpuResampSlc.h"
#include "gpuImageMode.h"

#include "isce/cuda/hip/hip_runtime_api.h"

using isce::cuda::core::gpuComplex;
using isce::cuda::core::gpuPoly2d;
using isce::cuda::core::gpuInterpolator;
using isce::cuda::core::gpuSinc2dInterpolator;
using isce::cuda::image::gpuImageMode;

#define THRD_PER_BLOCK 16 // Number of threads per block (should always %32==0)
#define SINC_ONE 9
#define SINC_HALF 4

__global__
void transformTile(const gpuComplex<float> *tile,
                   gpuComplex<float> *imgOut,
                   const float *rgOffTile,
                   const float *azOffTile,
                   const gpuPoly2d rgCarrier,
                   const gpuPoly2d azCarrier,
                   const gpuPoly2d doppler,
                   gpuImageMode mode,       // image mode for image to be resampled
                   gpuImageMode refMode,    // image mode for reference master image
                   gpuSinc2dInterpolator<gpuComplex<float>> *interp,
                   bool flatten,
                   int outWidth,
                   int outLength) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    printf("bdx%d, bix%d, tix%d\n", blockDim.x, blockIdx.x, threadIdx.x);
    printf("bdy%d, biy%d, tiy%d\n", blockDim.y, blockIdx.y, threadIdx.y);
    // check bounds
    if (i < outWidth && j < outLength) {
        int iArr = outWidth*i + j;

        gpuComplex<float> chip[SINC_ONE*SINC_ONE];

        // Unpack offsets
        const float azOff = azOffTile[iArr];
        const float rgOff = rgOffTile[iArr];
        printf("azOff %f | rgOff %f\n", azOff, rgOff);

        // Break into fractional and integer parts
        const int intAz = __float2int_rd(i + azOff);
        const int intRg = __float2int_rd(j + rgOff);
        const double fracAz = i + azOff - intAz;
        const double fracRg = j + rgOff - intRg;
       
        // Check bounds again
        bool intAzInBounds = !((intAz < SINC_HALF) || (intAz >= (outLength - SINC_HALF)));
        bool intRgInBounds = !((intRg < SINC_HALF) || (intRg >= (outWidth - SINC_HALF)));

        if (intAzInBounds && intRgInBounds) {
            // evaluate Doppler polynomial
            const double dop = doppler.eval(0, j) * 2 * M_PI / mode.prf;
            printf("dop %f\n", dop);

            // Doppler to be added back. Simultaneously evaluate carrier that needs to
            // be added back after interpolation
            double phase = (dop * fracAz) 
                + rgCarrier.eval(i + azOff, j + rgOff) 
                + azCarrier.eval(i + azOff, j + rgOff);

            printf("phase %f\n", phase);
            printf("azCarrier %f | rgCarrier %f\n", azCarrier.eval(i + azOff, j + rgOff), 
                                                    rgCarrier.eval(i + azOff, j + rgOff));
            // Flatten the carrier phase if requested
            if (flatten && refMode.isRefMode) {
                phase += ((4. * (M_PI / mode.wavelength)) * 
                    ((mode.startingRange - refMode.startingRange) 
                    + (j * (mode.rangePixelSpacing - refMode.rangePixelSpacing)) 
                    + (rgOff * mode.rangePixelSpacing))) + ((4.0 * M_PI 
                    * (refMode.startingRange + (j * refMode.rangePixelSpacing))) 
                    * ((1.0 / refMode.wavelength) - (1.0 / mode.wavelength)));
            }
            printf("phase %f\n", phase);
            // Modulate by 2*PI
            phase = fmod(phase, 2.0*M_PI);
            printf("phase %f\n", phase);
            
            // Read data chip without the carrier phases
            for (int ii = 0; ii < SINC_ONE; ++ii) {
                // Row to read from
                const int chipRow = intAz + ii - SINC_HALF;
                // Carrier phase
                const double phase = dop * (ii - 4.0);
                const gpuComplex<float> cval(cos(phase), -sin(phase));
                // Set the data values after removing doppler in azimuth
                for (int jj = 0; jj < SINC_ONE; ++jj) {
                    // Column to read from
                    const int chipCol = intRg + jj - SINC_HALF;
                    printf("intAz %d ii %d chipRow %d intRg %d jj %d chipCol %d\n", intAz, ii, chipRow, intRg, jj, chipCol);
                    chip[ii*SINC_ONE+jj] = tile[iArr] * cval;
                }
            }

            // Interpolate chip
            const gpuComplex<float> cval = interp->interpolate(
                SINC_HALF + fracRg + 1, SINC_HALF + fracAz + 1, chip, SINC_ONE, SINC_ONE
            );

            // Add doppler to interpolated value and save
            imgOut[iArr] = cval * gpuComplex<float>(cos(phase), sin(phase));
        }
    }
}


// Interpolate tile to perform transformation
void isce::cuda::image::
gpuTransformTile(isce::image::Tile<std::complex<float>> & tile,
               isce::io::Raster & outputSlc,
               isce::image::Tile<float> & rgOffTile,
               isce::image::Tile<float> & azOffTile,
               const isce::core::Poly2d & rgCarrier,
               const isce::core::Poly2d & azCarrier,
               const isce::core::Poly2d & doppler,
               isce::product::ImageMode mode,       // image mode for image to be resampled
               isce::product::ImageMode refMode,    // image mode for reference master image
               bool haveRefMode,
               int inLength, bool flatten) {

    // Cache geometry values
    const int inWidth = tile.width();
    const int outWidth = azOffTile.width();
    const int outLength = azOffTile.length();

    // Allocate valarray for output image block
    std::valarray<std::complex<float>> imgOut(outLength * outWidth);
    // Initialize to zeros
    imgOut = std::complex<float>(0.0, 0.0);

    // declare equivalent objects in device memory
    gpuComplex<float> *d_tile = NULL;
    gpuComplex<float> *d_imgOut = NULL;
    float *d_rgOffTile, *d_azOffTile;
    gpuPoly2d d_rgCarrier(rgCarrier);
    gpuPoly2d d_azCarrier(azCarrier);
    gpuImageMode d_mode(mode);
    gpuImageMode d_refMode;
    if (haveRefMode)
        gpuImageMode d_mode(refMode);
    gpuPoly2d d_doppler(doppler);
    gpuSinc2dInterpolator<gpuComplex<float>> d_interp(isce::core::SINC_LEN, isce::core::SINC_SUB);

    // allocate equivalent ofjects in device memory
    size_t nPixels = imgOut.size();
    size_t nComplexBytes = nPixels * sizeof(gpuComplex<float>);

    checkCudaErrors(hipMalloc(&d_tile, nPixels*sizeof(gpuComplex<float>)));
    checkCudaErrors(hipMalloc(&d_imgOut, nPixels*sizeof(gpuComplex<float>)));
    checkCudaErrors(hipMalloc(&d_azOffTile, nPixels*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_rgOffTile, nPixels*sizeof(float)));

    // copy objects to device memory
    checkCudaErrors(hipMemcpy(d_tile, &tile[tile.rowStart()], nPixels*sizeof(gpuComplex<float>), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_azOffTile, &azOffTile[azOffTile.rowStart()], nPixels*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rgOffTile, &rgOffTile[rgOffTile.rowStart()], nPixels*sizeof(float), hipMemcpyHostToDevice));

    // determine block layout
    dim3 block(THRD_PER_BLOCK, THRD_PER_BLOCK);
    const int nBlocks_x = (outWidth + THRD_PER_BLOCK - 1) / THRD_PER_BLOCK;
    const int nBlocks_y = (outLength + THRD_PER_BLOCK - 1) / THRD_PER_BLOCK;
    dim3 grid(nBlocks_x, nBlocks_y);

    // global call to transform
    //transformTile<<<grid,block>>>(d_tile, 
    transformTile<<<dim3(1),dim3(1)>>>(d_tile, 
                                d_imgOut, 
                                d_rgOffTile, 
                                d_azOffTile, 
                                d_rgCarrier, 
                                d_azCarrier, 
                                d_doppler, 
                                d_mode, 
                                d_refMode,
                                &d_interp,
                                flatten,
                                outWidth,
                                outLength); 

    // copy to host memory
    checkCudaErrors(hipMemcpy(&imgOut[0], d_imgOut, nPixels*sizeof(gpuComplex<float>), hipMemcpyDeviceToHost));

    // deallocate to device memory
    checkCudaErrors(hipFree(d_tile));
    checkCudaErrors(hipFree(d_imgOut));
    checkCudaErrors(hipFree(d_azOffTile));
    checkCudaErrors(hipFree(d_rgOffTile));
    
    // Write block of data
    outputSlc.setBlock(imgOut, 0, tile.rowStart(), outWidth, outLength);
}
