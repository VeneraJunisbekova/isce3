#include "hip/hip_runtime.h"
//-*- coding: utf-8 -*-
//
// Author: Liang Yu
// Copyright: 2018
#include <math.h>

// isce::core
#include "isce/core/Constants.h"
#include "isce/core/Interpolator.h"
#include "isce/core/Poly2d.h"

// isce::cuda::core
#include "isce/cuda/core/gpuComplex.h"
#include "isce/cuda/core/gpuInterpolator.h"
#include "isce/cuda/core/gpuPoly2d.h"

// isce::cuda::image
#include "gpuResampSlc.h"
#include "gpuImageMode.h"

#include "isce/cuda/hip/hip_runtime_api.h"

using isce::cuda::core::gpuComplex;
using isce::cuda::core::gpuPoly2d;
using isce::cuda::core::gpuInterpolator;
using isce::cuda::core::gpuSinc2dInterpolator;
using isce::cuda::image::gpuImageMode;

#define THRD_PER_BLOCK 512// Number of threads per block (should always %32==0)
#define SINC_ONE 9
#define SINC_HALF 4

__global__
void transformTile(const gpuComplex<float> *tile,
                   gpuComplex<float> *chip,
                   gpuComplex<float> *imgOut,
                   const float *rgOffTile,
                   const float *azOffTile,
                   const gpuPoly2d rgCarrier,
                   const gpuPoly2d azCarrier,
                   const gpuPoly2d doppler,
                   gpuImageMode mode,       // image mode for image to be resampled
                   gpuImageMode refMode,    // image mode for reference master image
                   gpuSinc2dInterpolator<gpuComplex<float>> interp,
                   bool flatten,
                   int outWidth,
                   int outLength) {

    int iTileOut = blockDim.x * blockIdx.x + threadIdx.x;
    int iChip = iTileOut * SINC_ONE * SINC_ONE;
    if (iTileOut < outWidth*outLength) {
        int i = iTileOut / outWidth;
        int j = iTileOut % outWidth;
        imgOut[iTileOut] = gpuComplex<float>(-1., 1.);

        // Unpack offsets
        const float azOff = azOffTile[iTileOut];
        const float rgOff = rgOffTile[iTileOut];

        // Break into fractional and integer parts
        const int intAz = __float2int_rd(i + azOff);
        const int intRg = __float2int_rd(j + rgOff);
        const double fracAz = i + azOff - intAz;
        const double fracRg = j + rgOff - intRg;
       
        // Check bounds again
        bool intAzInBounds = !((intAz < SINC_HALF) || (intAz >= (outLength - SINC_HALF)));
        bool intRgInBounds = !((intRg < SINC_HALF) || (intRg >= (outWidth - SINC_HALF)));

        if (intAzInBounds && intRgInBounds) {
            // evaluate Doppler polynomial
            const double dop = doppler.eval(0, j) * 2 * M_PI / mode.prf;

            // Doppler to be added back. Simultaneously evaluate carrier that needs to
            // be added back after interpolation
            double phase = (dop * fracAz) 
                + rgCarrier.eval(i + azOff, j + rgOff) 
                + azCarrier.eval(i + azOff, j + rgOff);

            // Flatten the carrier phase if requested
            if (flatten && refMode.isRefMode) {
                phase += ((4. * (M_PI / mode.wavelength)) * 
                    ((mode.startingRange - refMode.startingRange) 
                    + (j * (mode.rangePixelSpacing - refMode.rangePixelSpacing)) 
                    + (rgOff * mode.rangePixelSpacing))) + ((4.0 * M_PI 
                    * (refMode.startingRange + (j * refMode.rangePixelSpacing))) 
                    * ((1.0 / refMode.wavelength) - (1.0 / mode.wavelength)));
            }
            
            // Modulate by 2*PI
            phase = fmod(phase, 2.0*M_PI);
            
            // Read data chip without the carrier phases
            for (int ii = 0; ii < SINC_ONE; ++ii) {
                // Row to read from
                const int chipRow = intAz + ii - SINC_HALF;
                // Carrier phase
                const double phase = dop * (ii - 4.0);
                const gpuComplex<float> cval(cos(phase), -sin(phase));
                // Set the data values after removing doppler in azimuth
                for (int jj = 0; jj < SINC_ONE; ++jj) {
                    // Column to read from
                    const int chipCol = intRg + jj - SINC_HALF;
                    chip[iChip + ii*SINC_ONE+jj] = tile[chipRow*outWidth+chipCol] * cval;
                }
            }

            // Interpolate chip
            //const gpuComplex<float> cval(1., 1.);
            const gpuComplex<float> cval = interp.interpolate(
                SINC_HALF + fracRg + 1, SINC_HALF + fracAz + 1, &chip[iChip], SINC_ONE, SINC_ONE
            );

            // Add doppler to interpolated value and save
            imgOut[iTileOut] = cval * gpuComplex<float>(cos(phase), sin(phase));
        }
    }
}


// Interpolate tile to perform transformation
void isce::cuda::image::
gpuTransformTile(isce::image::Tile<std::complex<float>> & tile,
               isce::io::Raster & outputSlc,
               isce::image::Tile<float> & rgOffTile,
               isce::image::Tile<float> & azOffTile,
               const isce::core::Poly2d & rgCarrier,
               const isce::core::Poly2d & azCarrier,
               const isce::core::Poly2d & doppler,
               isce::product::ImageMode mode,       // image mode for image to be resampled
               isce::product::ImageMode refMode,    // image mode for reference master image
               bool haveRefMode,
               int inLength, bool flatten) {

    // Cache geometry values
    const int outWidth = azOffTile.width();
    const int outLength = azOffTile.length();

    // Allocate valarray for output image block
    std::valarray<std::complex<float>> imgOut(outLength * outWidth);
    // Initialize to zeros
    imgOut = std::complex<float>(0.0, 0.0);

    // declare equivalent objects in device memory
    gpuComplex<float> *d_tile;
    gpuComplex<float> *d_chip;
    gpuComplex<float> *d_imgOut;
    float *d_rgOffTile, *d_azOffTile;
    gpuPoly2d d_rgCarrier(rgCarrier);
    gpuPoly2d d_azCarrier(azCarrier);
    gpuImageMode d_mode(mode);
    gpuImageMode d_refMode;
    if (haveRefMode)
        gpuImageMode d_mode(refMode);
    gpuPoly2d d_doppler(doppler);

    // initialize interpolator
    gpuSinc2dInterpolator<gpuComplex<float>> d_interp(isce::core::SINC_LEN, isce::core::SINC_SUB);

    // allocate equivalent objects in device memory
    size_t nPixels = imgOut.size();
    size_t nTileBytes = nPixels * sizeof(gpuComplex<float>);
    size_t nChipBytes = nTileBytes * SINC_ONE * SINC_ONE;

    checkCudaErrors(hipMalloc(&d_tile, nTileBytes));
    checkCudaErrors(hipMalloc(&d_chip, nChipBytes));
    checkCudaErrors(hipMalloc(&d_imgOut, nTileBytes));
    checkCudaErrors(hipMalloc(&d_azOffTile, nPixels*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_rgOffTile, nPixels*sizeof(float)));

    // copy objects to device memory
    checkCudaErrors(hipMemcpy(d_tile, &tile[tile.rowStart()], nPixels*sizeof(gpuComplex<float>), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_azOffTile, &azOffTile[azOffTile.rowStart()], nPixels*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rgOffTile, &rgOffTile[rgOffTile.rowStart()], nPixels*sizeof(float), hipMemcpyHostToDevice));

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((nPixels+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    // global call to transform
    transformTile<<<grid, block>>>(d_tile, 
                                   d_chip,
                                   d_imgOut, 
                                   d_rgOffTile, 
                                   d_azOffTile, 
                                   d_rgCarrier, 
                                   d_azCarrier, 
                                   d_doppler, 
                                   d_mode, 
                                   d_refMode,
                                   d_interp,
                                   flatten,
                                   outWidth,
                                   outLength);

    // Check for any kernel errors
    checkCudaErrors(hipPeekAtLastError());

    // copy to host memory
    checkCudaErrors(hipMemcpy(&imgOut[0], d_imgOut, nPixels*sizeof(gpuComplex<float>), hipMemcpyDeviceToHost));

    // deallocate to device memory
    checkCudaErrors(hipFree(d_tile));
    checkCudaErrors(hipFree(d_chip));
    checkCudaErrors(hipFree(d_imgOut));
    checkCudaErrors(hipFree(d_azOffTile));
    checkCudaErrors(hipFree(d_rgOffTile));
    
    // Write block of data
    outputSlc.setBlock(imgOut, 0, tile.rowStart(), outWidth, outLength);
}
