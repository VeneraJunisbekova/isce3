#include "hip/hip_runtime.h"
// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// Source Author: Liang Yu
// Copyright 2019

#include <typeinfo>
#include <cstdio>
#include <string>

#include "gpuSignal.h"
#include "isce/cuda/core/gpuComplex.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include "isce/cuda/hip/hip_runtime_api.h"
#include "isce/cuda/helper_functions.h"

#define THRD_PER_BLOCK 1024 // Number of threads per block (should always %32==0)

using isce::cuda::signal::gpuSignal;
using isce::cuda::core::gpuComplex;

/** Constructor **/
template<class T>
gpuSignal<T>::
gpuSignal(hipfftType _type) {
    _cufft_type = _type;
    _plan_set = false;
    _d_data = NULL;
    _d_data_set = false;

    _n = new int[2];
    _inembed = new int[2];
    _onembed = new int[2];
}

/** Destructor **/
template<class T>
gpuSignal<T>::
~gpuSignal() {
    if (_plan_set)
        hipfftDestroy(_plan);

    if (_d_data_set)
        hipFree(_d_data);

    delete[] _n;
    delete[] _inembed;
    delete[] _onembed;
}

/**
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
rangeFFT(int ncolumns, int nrows)
{
    _configureRangeFFT(ncolumns, nrows);
    
    fftPlan(_rank, _n, _howmany,
            _inembed, _istride, _idist,
            _onembed, _ostride, _odist);
}

/**
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
azimuthFFT(int ncolumns, int nrows)
{
    _configureAzimuthFFT(ncolumns, nrows);
    
    fftPlan(_rank, _n, _howmany,
            _inembed, _istride, _idist,
            _onembed, _ostride, _odist);
}

/**
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
FFT2D(int ncolumns, int nrows)
{
    _n_elements = nrows * ncolumns;
    if (_plan_set) {
        hipfftDestroy(_plan);
    }

    checkCudaErrors(hipfftCreate(&_plan));
    size_t worksize;
    checkCudaErrors(hipfftMakePlan2d(_plan, nrows, ncolumns, _cufft_type, &worksize));
}

/**
*  @param[in] rank rank of the transform (1: for one dimensional and 2: for two dimensional transform)
*  @param[in] size size of each transform (ncols: for range FFT, nrows: for azimuth FFT)
*  @param[in] howmany number of FFT transforms for a block of data (nrows: for range FFT, ncols: for azimuth FFT)
*  @param[in] inembed
*  @param[in] istride
*  @param[in] idist
*  @param[in] onembed
*  @param[in] ostride
*  @param[in] odist
*/
template <class T>
void gpuSignal<T>::
fftPlan(int rank, int *n, int howmany,
        int *inembed, int istride, int idist,
        int *onembed, int ostride, int odist)
{
    if (_plan_set) {
        hipfftDestroy(_plan);
    }

    checkCudaErrors(hipfftCreate(&_plan));
    _plan_set = true;
    size_t worksize;
    checkCudaErrors(hipfftMakePlanMany(_plan, rank, n, 
                                      inembed, istride, idist, 
                                      onembed, ostride, odist, 
                                      _cufft_type, _howmany, &worksize));
}

/** @param[in] N the actual length of a signal
*   @param[in] fftLength next power of two 
*/
template <class T>
void gpuSignal<T>::
nextPowerOfTwo(size_t N, size_t &fftLength)
{
    for (size_t i = 0; i < 17; ++i) {
        fftLength = std::pow(2, i);
        if (fftLength >= N) {
            break;
        }
    }
}

/** @param[in] ncolumns number of columns
*   @param[in] nrows number of rows
*/
template <class T>
void gpuSignal<T>::
_configureRangeFFT(int ncolumns, int nrows)
{
    _rank = 1;
    _n[0] = ncolumns;

    _howmany = nrows;
    
    _inembed[0] = ncolumns;

    _istride = 1;
    _idist = ncolumns;
    
    _onembed[0] = ncolumns;

    _ostride = 1;
    _odist = ncolumns;

    _n_elements = nrows * ncolumns;

    _rows = nrows;
    _columns = ncolumns;
}

/** @param[in] ncolumns number of columns
*   @param[in] nrows number of rows
*/
template <class T>
void gpuSignal<T>::
_configureAzimuthFFT(int ncolumns, int nrows)
{
    _rank = 1;
    _n[0] = nrows;

    _howmany = ncolumns;

    _inembed[0] = nrows;

    _istride = ncolumns;
    _idist = 1;

    _onembed[0] = nrows;

    _ostride = ncolumns;
    _odist = 1;

    _n_elements = nrows * ncolumns;

    _rows = nrows;
    _columns = ncolumns;
}

template<class T>
void gpuSignal<T>::
zeroDeviceData()
{
    if (!_d_data_set) {
        size_t input_size = _n_elements*sizeof(T)*2;
        // allocate input
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&_d_data), input_size));
        // zero out allocated data
        checkCudaErrors(hipMemset(_d_data, 0, input_size));
        _d_data_set = true;
    }
}

template<class T>
void gpuSignal<T>::
dataToDevice(std::complex<T> *input)
{
    if (!_d_data_set) {
        size_t input_size = _n_elements*sizeof(T)*2;
        // allocate input
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&_d_data), input_size));
        // copy input
        checkCudaErrors(hipMemcpy(_d_data, input, input_size, hipMemcpyHostToDevice));
        _d_data_set = true;
    }
}

template<class T>
void gpuSignal<T>::
dataToDevice(std::valarray<std::complex<T>> &input)
{
    if (!_d_data_set) {
        size_t input_size = input.size()*sizeof(T)*2;
        // allocate input
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&_d_data), input_size));
        // copy input
        checkCudaErrors(hipMemcpy(_d_data, &input[0], input_size, hipMemcpyHostToDevice));
        _d_data_set = true;
    }
}

template<class T>
void gpuSignal<T>::
dataToHost(std::complex<T> *output)
{
    if (_d_data_set) {
        size_t output_size = _n_elements*sizeof(T)*2;
        // copy output 
        checkCudaErrors(hipMemcpy(output, _d_data, output_size, hipMemcpyDeviceToHost));
    }
}

template<class T>
void gpuSignal<T>::
dataToHost(std::valarray<std::complex<T>> &output)
{
    if (_d_data_set) {
        size_t output_size = _n_elements*sizeof(T)*2;
        // copy output 
        checkCudaErrors(hipMemcpy(&output[0], _d_data, output_size, hipMemcpyDeviceToHost));
    }
}

/** unnormalized forward transform
*/
template<class T>
void gpuSignal<T>::
forwardC2C()
{
    if (_plan_set && _d_data_set)
        // transform
        checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(_d_data),
                                    reinterpret_cast<hipfftComplex *>(_d_data),
                                    HIPFFT_FORWARD));
}

/** unnormalized forward transform
*/
template<class T>
void gpuSignal<T>::
forwardZ2Z()
{
    if (_plan_set && _d_data_set)
        // transform
        checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(_d_data),
                                    reinterpret_cast<hipfftDoubleComplex *>(_d_data),
                                    HIPFFT_FORWARD));
}

template<>
void gpuSignal<float>::
forward()
{
    forwardC2C();
}

template<>
void gpuSignal<double>::
forward()
{
    forwardZ2Z();
}


/** unnormalized forward transform
*/
template<>
void gpuSignal<float>::
forwardDevMem(float *input, float *output)
{
    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(input),
                reinterpret_cast<hipfftComplex *>(output),
                HIPFFT_FORWARD));
}

/** unnormalized forward transform
*/
template<>
void gpuSignal<double>::
forwardDevMem(double *input, double *output)
{
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(input),
                reinterpret_cast<hipfftDoubleComplex *>(output),
                HIPFFT_FORWARD));
}

template<class T>
void gpuSignal<T>::
forwardDevMem(T *dataInPlace) {
    forwardDevMem(dataInPlace, dataInPlace);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardC2C(std::complex<T> *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardC2C(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardZ2Z(std::complex<T> *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

template<class T>
void gpuSignal<T>::
forwardZ2Z(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void gpuSignal<T>::
forwardD2Z(T *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T);
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform (implicitly forward)
    checkCudaErrors(hipfftExecD2Z(_plan, reinterpret_cast<hipfftDoubleReal *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_output)));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

template<>
void gpuSignal<float>::
forward(std::complex<float> *input, std::complex<float> *output)
{
    forwardC2C(input, output);
}

template<>
void gpuSignal<double>::
forward(std::complex<double> *input, std::complex<double> *output)
{
    forwardZ2Z(input, output);
}

template<>
void gpuSignal<float>::
forward(std::valarray<std::complex<float>> &input, std::valarray<std::complex<float>> &output)
{
    forwardC2C(input, output);
}

template<>
void gpuSignal<double>::
forward(std::valarray<std::complex<double>> &input, std::valarray<std::complex<double>> &output)
{
    forwardZ2Z(input, output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<>
void gpuSignal<float>::
inverseC2C()
{
    if (_plan_set && _d_data_set)
        // transform
        checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(_d_data),
                                    reinterpret_cast<hipfftComplex *>(_d_data),
                                    HIPFFT_BACKWARD));
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<>
void gpuSignal<double>::
inverseZ2Z()
{
    if (_plan_set && _d_data_set)
        // transform
        checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(_d_data),
                                    reinterpret_cast<hipfftDoubleComplex *>(_d_data),
                                    HIPFFT_BACKWARD));
}

template<>
void gpuSignal<float>::
inverse()
{
    inverseC2C();
}

template<>
void gpuSignal<double>::
inverse()
{
    inverseZ2Z();
}

/** unnormalized inverse transform
*/
template<>
void gpuSignal<float>::
inverseDevMem(float *input, float *output)
{
    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(input),
                reinterpret_cast<hipfftComplex *>(output),
                HIPFFT_BACKWARD));
}

/** unnormalized inverse transform
*/
template<>
void gpuSignal<double>::
inverseDevMem(double *input, double *output)
{
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(input),
                reinterpret_cast<hipfftDoubleComplex *>(output),
                HIPFFT_BACKWARD));
}

template<class T>
void gpuSignal<T>::
inverseDevMem(T *dataInPlace) {
    inverseDevMem(dataInPlace, dataInPlace);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void gpuSignal<T>::
inverseC2C(std::complex<T> *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void gpuSignal<T>::
inverseC2C(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void gpuSignal<T>::
inverseZ2Z(std::complex<T> *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

template<class T>
void gpuSignal<T>::
inverseZ2Z(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized inverse transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void gpuSignal<T>::
inverseZ2D(std::complex<T> *input, T *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T);

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform (implicitly inverse)
    checkCudaErrors(hipfftExecZ2D(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleReal *>(d_output)));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

template<>
void gpuSignal<double>::
inverse(std::complex<double> *input, std::complex<double> *output)
{
    inverseZ2Z(input, output);
}

template<>
void gpuSignal<float>::
inverse(std::valarray<std::complex<float>> &input, std::valarray<std::complex<float>> &output)
{
    inverseC2C(input, output);
}


template<>
void gpuSignal<float>::
upsample(std::valarray<std::complex<float>> &input,
          std::valarray<std::complex<float>> &output,
          int row, int nfft, int upsampleFactor,
          std::valarray<std::complex<float>> &shiftImpact)
{
    size_t input_size = _n_elements*sizeof(float)*2;
    size_t output_size = upsampleFactor * _n_elements*sizeof(float)*2;

    // allocate device memory 
    float *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    float *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // forward transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_FORWARD));

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((input_size+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    // shift data prior to upsampling transform
    if (shiftImpact.size() == output.size()) {
        float *d_shift_impact;
        size_t shift_size = shiftImpact.size()*sizeof(float)*2;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_shift_impact), shift_size));
        checkCudaErrors(hipMemcpy(d_shift_impact, &shiftImpact[0], shift_size, hipMemcpyHostToDevice));
        rangeShiftImpactMult_g<float><<<grid, block>>>(
                reinterpret_cast<gpuComplex<float> *>(d_input), reinterpret_cast<gpuComplex<float> *>(d_output), 
                reinterpret_cast<gpuComplex<float> *>(d_shift_impact),
                _rows, _columns, upsampleFactor*_columns);
        hipFree(d_shift_impact);
    }
    else
        rangeShift_g<float><<<grid, block>>>(
                reinterpret_cast<gpuComplex<float> *>(d_input), reinterpret_cast<gpuComplex<float> *>(d_output), 
                _rows, _columns, upsampleFactor*_columns);

    // set inverse transform
    rangeFFT(upsampleFactor*nfft, 1);

    // inverse transformation:693
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_output),
                                reinterpret_cast<hipfftComplex *>(d_output),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_output, output_size, hipMemcpyDeviceToHost));
}


template<>
void gpuSignal<float>::
upsample(std::valarray<std::complex<float>> &input,
          std::valarray<std::complex<float>> &output,
          int row, int nfft, int upsampleFactor)
{
    std::valarray<std::complex<float>> shiftImpact(0);

    upsample(input, output,
            row, nfft, upsampleFactor,
            shiftImpact);
}


/*
template<>
void gpuSignal<double>::
upsample(std::valarray<std::complex<double>> &input,
              std::valarray<std::complex<double>> &output,
              int row, int nfft, int upsampleFactor,
              std::valarray<std::complex<double>> &shiftImpact)
{
    size_t input_size = _n_elements*sizeof(double)*2;
    size_t output_size = upsampleFactor * _n_elements*sizeof(double)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, $input[0], input_size, hipMemcpyHostToDevice));

    // forward transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_FORWARD));

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((nOutPixels+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    if (shiftImpact.size() == output.size()) {
        gpuComplex<double> *d_shift_impact;
        size_t shift_size = shiftImpact.size()*sizeof(double)*2;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_shift_impact), shift_size));
        checkCudaErrors(hipMemcpy(d_shift_impact, &shiftImpact[0], shift_size, hipMemcpyHostToDevice));
        rangeShiftImpactMult_g<gpuComplex<double>><<<grid, block>>>(
                d_input, d_output, 
                d_shift_impact,
                _rows, _columns, upsampleFactor*_columns);
        hipFree(d_shift_impact);
    }
    else
        rangeShift_g<gpuComplex<double>><<<grid, block>>>(
                d_input, d_output, 
                _rows, _columns, upsampleFactor*_columns);

    // set inverse transform
    this.rangeFFT(oversample*nfft, 1);

    // inverse transformation:693
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftComplex *>(d_output),
                                reinterpret_cast<hipfftComplex *>(d_output),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_output, output_size, hipMemcpyDeviceToHost));
}
*/

/** 1D shift (in range only)
TODO move to GPU to eliminate
*  lo res N x M_lo copied into hi res N x M_hi
*  where M_hi = f_upsample x M_lo
*  @param[in]
*  @param[out]
*  @param[in]
*  @param[in]
*  @param[in]
*/
template<class T>
void shift(std::valarray<std::complex<T>> &spectrum,
           std::valarray<std::complex<T>> &spectrumShifted,
           int rows, int nfft, int columns)
{
    //spectrum /=nfft;
    //shift the spectrum
    // The spectrum has values from begining to nfft index for each line. We want
    // to put the spectrum in correct ouput locations such that the spectrum of
    // the upsampled data has values from 0 to nfft/2 and from upsampleFactor*nfft - nfft/2 to the end.
    // For a 1D example:
    //      spectrum = [1,2,3,4,5,6,0,0,0,0,0,0]
    //  becomes:
    //      spectrumShifted = [1,2,3,0,0,0,0,0,0,4,5,6]
    size_t right_offset = columns - nfft/2;
    for (size_t i_row = 0; i_row < rows; ++i_row) {
        size_t row_offset_lo = i_row * nfft;
        size_t row_offset_hi = i_row * columns;
        // copy left side of lo res to left side of hi res
        spectrumShifted[std::slice(row_offset_hi, nfft/2, 1)] = spectrum[std::slice(row_offset_lo, nfft/2, 1)];
        // copy right side of lo res to right side of hi res
        spectrumShifted[std::slice(row_offset_hi + right_offset, nfft/2, 1)] = spectrum[std::slice(row_offset_lo+nfft/2, nfft/2, 1)];
    }
}
/**
    recast inputs to either hipfftComplex or hipfftDoubleComplex
*/
template<class T>
__global__ void rangeShift_g(gpuComplex<T> *data_lo_res, gpuComplex<T> *data_hi_res, int n_rows, int n_cols_lo, int n_cols_hi)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int i_col = i % n_cols_lo;

    if (i < n_cols_lo * n_rows) {
        if (i_col < n_cols_lo / 2)
            data_hi_res[i*n_cols_hi + i_col] = data_lo_res[i]; 
        else
            data_hi_res[(i+1) * n_cols_hi - (n_cols_lo-i_col)] = data_lo_res[i];
    }
}

/**
    recast inputs to either hipfftComplex or hipfftDoubleComplex
*/
template<class T>
__global__ void rangeShiftImpactMult_g(gpuComplex<T> *data_lo_res, gpuComplex<T> *data_hi_res, gpuComplex<T> *shiftImpact, int n_rows, int n_cols_lo, int n_cols_hi)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int i_col = i % n_cols_lo;

    if (i < n_cols_lo * n_rows) {
        if (i_col < n_cols_lo / 2)
            data_hi_res[i*n_cols_hi + i_col] = data_lo_res[i] * shiftImpact[i]; 
        else
            data_hi_res[(i+1) * n_cols_hi - (n_cols_lo-i_col)] = data_lo_res[i] * shiftImpact[i];
    }
}

template<class T>
void upsample(isce::cuda::signal::gpuSignal<T> &fwd,
        isce::cuda::signal::gpuSignal<T> &inv,
        T *input,
        T *output)
{
    fwd.forward(input);

    // shift data prior to upsampling transform
    int num_blocks = max(fwd.getNumElements() / 1024, 1);
    auto lo_res_ptr = reinterpret_cast<gpuComplex<T> *>(input);
    auto hi_res_ptr = reinterpret_cast<gpuComplex<T> *>(output);
    rangeShift_g<T><<<num_blocks, 1024>>>(
            reinterpret_cast<gpuComplex<T> *>(lo_res_ptr), 
            reinterpret_cast<gpuComplex<T> *>(hi_res_ptr), 
            fwd.getRows(), fwd.getColumns(), inv.getColumns());

    inv.inverse(output);
}

template<class T>
void upsample(isce::cuda::signal::gpuSignal<T> &fwd,
        isce::cuda::signal::gpuSignal<T> &inv,
        T *input,
        T *output,
        T *shiftImpact)
{
    fwd.forward(input);

    // shift data prior to upsampling transform
    int num_blocks = max(fwd.getNumElements() / 1024, 1);
    auto lo_res_ptr = reinterpret_cast<gpuComplex<T> *>(input);
    auto hi_res_ptr = reinterpret_cast<gpuComplex<T> *>(output);
    rangeShiftImpactMult_g<T><<<num_blocks, 1024>>>(
            reinterpret_cast<gpuComplex<T> *>(lo_res_ptr), 
            reinterpret_cast<gpuComplex<T> *>(hi_res_ptr), 
            reinterpret_cast<gpuComplex<T> *>(shiftImpact),
            fwd.getRows(), fwd.getColumns(), inv.getColumns());

    inv.inverse(output);
}

void upsampleC2C(isce::cuda::signal::gpuSignal<float> &fwd,
                 isce::cuda::signal::gpuSignal<float> &inv,
                 std::valarray<std::complex<float>> &shiftImpact)
{
    // transform
    fwd.forwardC2C();

    // shift data prior to upsampling transform
    int num_blocks = max(fwd.getNumElements() / 1024, 1);
    auto lo_res_ptr = reinterpret_cast<gpuComplex<float> *>(fwd.getDevicePtr());
    auto hi_res_ptr = reinterpret_cast<gpuComplex<float> *>(inv.getDevicePtr());
    if (shiftImpact.size() == fwd.getNumElements()) {
        gpuComplex<float> *d_shift_impact;
        size_t shift_size = shiftImpact.size()*sizeof(float)*2;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_shift_impact), shift_size));
        checkCudaErrors(hipMemcpy(d_shift_impact, &shiftImpact[0], shift_size, hipMemcpyHostToDevice));
        rangeShiftImpactMult_g<float><<<num_blocks, 1024>>>(
                reinterpret_cast<gpuComplex<float> *>(lo_res_ptr), reinterpret_cast<gpuComplex<float> *>(hi_res_ptr), 
                reinterpret_cast<gpuComplex<float> *>(d_shift_impact),
                fwd.getRows(), fwd.getColumns(), inv.getColumns());
        hipFree(d_shift_impact);
    }
    else
        rangeShift_g<float><<<num_blocks, 1024>>>(
                reinterpret_cast<gpuComplex<float> *>(lo_res_ptr), reinterpret_cast<gpuComplex<float> *>(hi_res_ptr), 
                fwd.getRows(), fwd.getColumns(), inv.getColumns());

    // transform
    inv.inverseC2C();
}

void upsampleC2C(isce::cuda::signal::gpuSignal<float> &fwd,
                 isce::cuda::signal::gpuSignal<float> &inv,
                 std::valarray<std::complex<float>> &input,
                 std::valarray<std::complex<float>> &output,
                 std::valarray<std::complex<float>> &shiftImpact)
{
    // temporary storage for the spectrum before and after the shift
    std::valarray<std::complex<float>> spectrum(input.size());
    std::valarray<std::complex<float>> spectrumShifted(output.size());

    spectrumShifted = std::complex<float> (0.0,0.0);

    // transform
    fwd.forwardC2C(input, spectrum);

    // shift data prior to upsampling transform on CPU
    shift<float>(spectrum, spectrumShifted, fwd.getRows(), fwd.getColumns(), inv.getColumns());

    // multiply the shiftImpact (a linear phase is frequency domain
    // equivalent to a shift in time domain) by the spectrum
    if (spectrumShifted.size() == shiftImpact.size())
        spectrumShifted *= shiftImpact;

    // transform with upsampled spectrum
    inv.inverseC2C(spectrumShifted, output);
}

void upsampleZ2Z(isce::cuda::signal::gpuSignal<double> &lo_res,
                 isce::cuda::signal::gpuSignal<double> &hi_res,
                 std::valarray<std::complex<double>> &shiftImpact)
{
    // transform
    lo_res.forwardZ2Z();

    // shift data prior to upsampling transform
    int num_blocks = max(lo_res.getNumElements() / 1024, 1);
    auto lo_res_ptr = reinterpret_cast<gpuComplex<double> *>(lo_res.getDevicePtr());
    auto hi_res_ptr = reinterpret_cast<gpuComplex<double> *>(hi_res.getDevicePtr());
    if (shiftImpact.size() == lo_res.getNumElements()) {
        gpuComplex<double> *d_shift_impact;
        size_t shift_size = shiftImpact.size()*sizeof(double)*2;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_shift_impact), shift_size));
        checkCudaErrors(hipMemcpy(d_shift_impact, &shiftImpact[0], shift_size, hipMemcpyHostToDevice));
        rangeShiftImpactMult_g<double><<<num_blocks, 1024>>>(
                reinterpret_cast<gpuComplex<double> *>(lo_res_ptr), reinterpret_cast<gpuComplex<double> *>(hi_res_ptr), 
                reinterpret_cast<gpuComplex<double> *>(d_shift_impact),
                lo_res.getRows(), lo_res.getColumns(), hi_res.getColumns());
        hipFree(d_shift_impact);
    }
    else
        rangeShift_g<double><<<num_blocks, 1024>>>(
                reinterpret_cast<gpuComplex<double> *>(lo_res_ptr), reinterpret_cast<gpuComplex<double> *>(hi_res_ptr), 
                lo_res.getRows(), lo_res.getColumns(), hi_res.getColumns());

    // transform
    hi_res.inverseZ2Z();
}

void upsampleZ2Z(isce::cuda::signal::gpuSignal<double> &fwd,
                 isce::cuda::signal::gpuSignal<double> &inv,
                 std::valarray<std::complex<double>> &input,
                 std::valarray<std::complex<double>> &output,
                 std::valarray<std::complex<double>> &shiftImpact)
{
    // temporary storage for the spectrum before and after the shift
    std::valarray<std::complex<double>> spectrum(input.size());
    std::valarray<std::complex<double>> spectrumShifted(output.size());

    spectrum = std::complex<double> (0.0,0.0);
    spectrumShifted = std::complex<double> (0.0,0.0);

    // transform
    fwd.forwardZ2Z(input, spectrum);

    // shift data prior to upsampling transform on CPU
    shift<double>(spectrum, spectrumShifted, fwd.getRows(), fwd.getColumns(), inv.getColumns());

    // multiply the shiftImpact (a linear phase is frequency domain
    // equivalent to a shift in time domain) by the spectrum
    if (spectrumShifted.size() == shiftImpact.size())
        spectrumShifted *= shiftImpact;

    // transform
    inv.inverseZ2Z(spectrumShifted, output);
}

/*
 each template parameter needs it's own declaration here
 */
template class gpuSignal<float>;
template class gpuSignal<double>;

template __global__ void 
rangeShift_g<float>(gpuComplex<float> *data_lo_res, gpuComplex<float> *data_hi_res, 
        int n_rows, int n_cols_lo, int n_cols_hi);

template __global__ void 
rangeShift_g<double>(gpuComplex<double> *data_lo_res, gpuComplex<double> *data_hi_res, 
        int n_rows, int n_cols_lo, int n_cols_hi);

template __global__ void 
rangeShiftImpactMult_g<float>(gpuComplex<float> *data_lo_res, gpuComplex<float> *data_hi_res, 
        gpuComplex<float> *impact_shift, 
        int n_rows, int n_cols_lo, int n_cols_hi);

template __global__ void 
rangeShiftImpactMult_g<double>(gpuComplex<double> *data_lo_res, gpuComplex<double> *data_hi_res, 
        gpuComplex<double> *impact_shift, 
        int n_rows, int n_cols_lo, int n_cols_hi);
