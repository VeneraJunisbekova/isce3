#include "hip/hip_runtime.h"
// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// Source Author: Liang Yu
// Copyright 2019

#include <typeinfo>
#include <cstdio>
#include <string>

#include "gpuSignal.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include "isce/cuda/hip/hip_runtime_api.h"
#include "isce/cuda/helper_functions.h"

using isce::cuda::signal::gpuSignal;


/** Destructor **/
template<class T>
gpuSignal<T>::
~gpuSignal() {
    hipfftDestroy(_plan);
}

/**
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
rangeFFT(int ncolumns, int nrows)
{
    _configureRangeFFT(ncolumns, nrows);
    
    fftPlan(_rank, _n, _howmany,
            _inembed, _istride, _idist,
            _onembed, _ostride, _odist);
}

/**
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
azimuthFFT(int ncolumns, int nrows)
{
    _configureAzimuthFFT(ncolumns, nrows);
    
    fftPlan(_rank, _n, _howmany,
            _inembed, _istride, _idist,
            _onembed, _ostride, _odist);
}

/**
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
FFT2D(int ncolumns, int nrows)
{
    _n_elements = nrows * ncolumns;
    checkCudaErrors(hipfftCreate(&_plan));
    size_t worksize;
    checkCudaErrors(hipfftMakePlan2d(_plan, nrows, ncolumns, _cufft_type, &worksize));
}

/**
*  @param[in] rank rank of the transform (1: for one dimensional and 2: for two dimensional transform)
*  @param[in] size size of each transform (ncols: for range FFT, nrows: for azimuth FFT)
*  @param[in] howmany number of FFT transforms for a block of data (nrows: for range FFT, ncols: for azimuth FFT)
*  @param[in] inembed
*  @param[in] istride
*  @param[in] idist
*  @param[in] onembed
*  @param[in] ostride
*  @param[in] odist
*/
template <class T>
void gpuSignal<T>::
fftPlan(int rank, int *n, int howmany,
        int *inembed, int istride, int idist,
        int *onembed, int ostride, int odist)
{
    checkCudaErrors(hipfftCreate(&_plan));
    size_t worksize;
    checkCudaErrors(hipfftMakePlanMany(_plan, rank, n, 
                                      inembed, istride, idist, 
                                      onembed, ostride, odist, 
                                      _cufft_type, _howmany, &worksize));
}

/** @param[in] N the actual length of a signal
*   @param[in] fftLength next power of two 
*/
template <class T>
void gpuSignal<T>::
nextPowerOfTwo(size_t N, size_t &fftLength)
{
    for (size_t i = 0; i < 17; ++i) {
        fftLength = std::pow(2, i);
        if (fftLength >= N) {
            break;
        }
    }
}

/** @param[in] ncolumns number of columns
*   @param[in] nrows number of rows
*/
template <class T>
void gpuSignal<T>::
_configureRangeFFT(int ncolumns, int nrows)
{
    _rank = 1;
    _n = new int[1];
    _n[0] = ncolumns;

    _howmany = nrows;
    
    _inembed = new int[1];
    _inembed[0] = ncolumns;

    _istride = 1;
    _idist = ncolumns;
    
    _onembed = new int[1];
    _onembed[0] = ncolumns;

    _ostride = 1;
    _odist = ncolumns;
}

/** @param[in] ncolumns number of columns
*   @param[in] nrows number of rows
*/
template <class T>
void gpuSignal<T>::
_configureAzimuthFFT(int ncolumns, int nrows)
{
    _rank = 1;
    _n = new int[1];
    _n[0] = nrows;

    _howmany = ncolumns;

    _inembed = new int[1];
    _inembed[0] = nrows;

    _istride = ncolumns;
    _idist = 1;

    _onembed = new int[1];
    _onembed[0] = nrows;

    _ostride = ncolumns;
    _odist = 1;
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void isce::cuda::signal::gpuSignal<T>::
forwardC2C(std::complex<T> *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void isce::cuda::signal::gpuSignal<T>::
forwardC2C(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void isce::cuda::signal::gpuSignal<T>::
forwardZ2Z(std::complex<T> *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

template<class T>
void isce::cuda::signal::gpuSignal<T>::
forwardZ2Z(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void isce::cuda::signal::gpuSignal<T>::
inverseC2C(std::complex<T> *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void isce::cuda::signal::gpuSignal<T>::
inverseC2C(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void isce::cuda::signal::gpuSignal<T>::
inverseZ2Z(std::complex<T> *input, std::complex<T> *output)
{
    size_t input_size = _n_elements*sizeof(T)*2;
    size_t output_size = _n_elements*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(output, d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

template<class T>
void isce::cuda::signal::gpuSignal<T>::
inverseZ2Z(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecZ2Z(_plan, reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                reinterpret_cast<hipfftDoubleComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/*
 each template parameter needs it's own declaration here
 */
template class gpuSignal<float>;
template class gpuSignal<double>;
