// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// Source Author: Liang Yu
// Copyright 2019

#include "gpuSignal.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include "isce/cuda/hip/hip_runtime_api.h"
#include "isce/cuda/helper_functions.h"

using isce::cuda::signal::gpuSignal;


/** Destructor **/
template<class T>
gpuSignal<T>::
~gpuSignal() {
    hipfftDestroy(_plan);
}

/**
*  @param[in] signal input block of data
*  @param[out] spectrum output block of spectrum
*  @param[in] ncolumns number of columns of the block of data
*  @param[in] nrows number of rows of the block of data
 */
template<class T>
void gpuSignal<T>::
forwardRangeFFT(int ncolumns, int nrows)
                
{

    _configureRangeFFT(ncolumns, nrows);
    
    fftPlanForward(_rank, _n, _howmany,
                    _inembed, _istride, _idist,
                    _onembed, _ostride, _odist);

}

/**
*  @param[in] rank rank of the transform (1: for one dimensional and 2: for two dimensional transform)
*  @param[in] size size of each transform (ncols: for range FFT, nrows: for azimuth FFT)
*  @param[in] howmany number of FFT transforms for a block of data (nrows: for range FFT, ncols: for azimuth FFT)
*  @param[in] inembed
*  @param[in] istride
*  @param[in] idist
*  @param[in] onembed
*  @param[in] ostride
*  @param[in] odist
*/
template <class T>
void gpuSignal<T>::
fftPlanForward(int rank, int *n, int howmany,
                int *inembed, int istride, int idist,
                int *onembed, int ostride, int odist)
{
    _cufft_type = HIPFFT_C2C;
    checkCudaErrors(hipfftCreate(&_plan));
    size_t worksize;
    checkCudaErrors(hipfftMakePlanMany(_plan, rank, n, inembed,
                                      istride, idist, onembed, ostride, 
                                      odist, _cufft_type, _howmany, &worksize));

    //checkCudaErrors(hipfftCreate(&_plan));checkCudaErrors(hipfftMakePlan2d(_plan, n[0], n[1], _cufft_type, worksize));
}

template <class T>
void gpuSignal<T>::dbgTodos(int n, 
                           std::valarray<std::complex<T>> &input, 
                           std::valarray<std::complex<T>> &output)
{
    _cufft_type = HIPFFT_C2C;
    checkCudaErrors(hipfftPlan1d(&_plan, n, _cufft_type, 1));

    size_t input_size = input.size()*sizeof(std::complex<T>);
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input from host to device
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // temp for input
    float *h_temp = reinterpret_cast<float *>(malloc(input_size));
    checkCudaErrors(hipMemcpy(h_temp, d_input, input_size, hipMemcpyDeviceToHost));
    std::fstream fin;
    fin.open("data_in.bin", std::ios::out | std::ios::binary);
    fin.write((char *)h_temp, input_size);
    fin.close();

    // transform in place
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output from device to host
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    // redundant write out
    std::fstream fredun;
    fredun.open("data_out_redundant.bin", std::ios::out | std::ios::binary);
    fredun.write((char *)&output[0], input_size);
    fredun.close();
    
    // temp for output
    checkCudaErrors(hipMemcpy(h_temp, d_input, input_size, hipMemcpyDeviceToHost));
    std::fstream fout;
    fout.open("data_out.bin", std::ios::out | std::ios::binary);
    fout.write((char *)h_temp, input_size);
    fout.close();

    free(h_temp);
    
    hipFree(d_input);
    hipFree(d_output);
}

/** @param[in] ncolumns number of columns
*   @param[in] nrows number of rows
*/
template <class T>
void gpuSignal<T>::
_configureRangeFFT(int ncolumns, int nrows)
{
    _rank = 1;                  // dimensionality of transform
    _n = new int[1];
    _n[0] = ncolumns;

    _howmany = nrows;
    
    _inembed = new int[1];
    _inembed[0] = ncolumns;

    _istride = 1;
    _idist = ncolumns;
    
    _onembed = new int[1];
    _onembed[0] = ncolumns;

    _ostride = 1;
    _odist = ncolumns;
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void isce::cuda::signal::gpuSignal<T>::
forward(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(std::complex<T>);
    //size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_FORWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/** unnormalized forward transform
*  @param[in] input block of data
*  @param[out] output block of spectrum
*/
template<class T>
void isce::cuda::signal::gpuSignal<T>::
inverse(std::valarray<std::complex<T>> &input, std::valarray<std::complex<T>> &output)
{
    size_t input_size = input.size()*sizeof(std::complex<T>);
    //size_t input_size = input.size()*sizeof(T)*2;
    size_t output_size = output.size()*sizeof(T)*2;

    // allocate device memory 
    T *d_input;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), input_size));
    T *d_output;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_output), output_size));

    // copy input
    checkCudaErrors(hipMemcpy(d_input, &input[0], input_size, hipMemcpyHostToDevice));

    // transform
    checkCudaErrors(hipfftExecC2C(_plan, reinterpret_cast<hipfftComplex *>(d_input),
                                reinterpret_cast<hipfftComplex *>(d_input),
                                HIPFFT_BACKWARD));

    // copy output
    checkCudaErrors(hipMemcpy(&output[0], d_input, input_size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}

/**
*  @param[in] rank rank of the transform (1: for one dimensional and 2: for two dimensional transform)
*  @param[in] size size of each transform (ncols: for range FFT, nrows: for azimuth FFT)
*  @param[in] howmany number of FFT transforms for a block of data (nrows: for range FFT, ncols: for azimuth FFT)
*  @param[in] inembed
*  @param[in] istride
*  @param[in] idist
*  @param[in] onembed
*  @param[in] ostride
*  @param[in] odist
template <class T>
void
gpuSignal<T>::
fftPlanBackward(int rank, int *n, int howmany,
                int *inembed, int istride, int idist,
                int *onembed, int ostride, int odist)
{
    size_t worksize;
    checkCudaErrors(hipfftCreate(&_plan_inv));
    checkCudaErrors(hipfftMakePlanMany(_plan_inv, rank, n, inembed,
                                      istride, idist, onembed, ostride, 
                                      odist, _cufft_type, 1, worksize)
}

*/
/*
 each template parameter needs it's own declaration here
 */
template class gpuSignal<float>;
