#include "hip/hip_runtime.h"
// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// Source Author: Liang Yu
// Copyright 2019

#include "gpuFilter.h"
#include "isce/io/Raster.h"

#include "isce/cuda/hip/hip_runtime_api.h"
#include "isce/cuda/helper_functions.h"

#define THRD_PER_BLOCK 1024 // Number of threads per block (should always %32==0)

using isce::cuda::signal::gpuFilter;

template<class T>
gpuFilter<T>::~gpuFilter()
{
    if (_filter_set) {
        hipFree(_d_filter);
    }
}

// do all calculations in place with data stored on device within signal
template<class T>
void gpuFilter<T>::
filter(gpuSignal<T> &signal)
{
    signal.forward();

    auto n_signal_elements = signal.getNumElements();

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((n_signal_elements+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    filter_g<<<grid, block>>>(reinterpret_cast<gpuComplex<T> *>(signal.getDevicePtr()), 
            reinterpret_cast<gpuComplex<T> *>(&_d_filter), 
            n_signal_elements);

    signal.inverse();
}


// pass in device pointer to filter on
template<class T>
void gpuFilter<T>::
filter(gpuComplex<T> *data)
{
    _signal.forwardDevMem(reinterpret_cast<T *>(data));

    auto n_signal_elements = _signal.getNumElements();

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((n_signal_elements+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    filter_g<<<grid, block>>>(data, 
            reinterpret_cast<gpuComplex<T> *>(_d_filter), 
            n_signal_elements);

    _signal.inverseDevMem(reinterpret_cast<T *>(data));
}


// pass in host memory to copy to device to be filtered
// interim spectrum is saved as well
template<class T>
void gpuFilter<T>::
filter(std::valarray<std::complex<T>> &signal,
        std::valarray<std::complex<T>> &spectrum)
{
    _signal.dataToDevice(signal);
    _signal.forward();

    // save spectrum
    _signal.dataToHost(spectrum);

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((signal.size()+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    filter_g<<<grid, block>>>(reinterpret_cast<gpuComplex<T> *>(_signal.getDevicePtr()), 
            reinterpret_cast<gpuComplex<T> *>(&_d_filter), 
            signal.size());

    _signal.inverse();

    // copy signal to host
    _signal.dataToHost(signal);
}

template<class T>
void gpuFilter<T>::
cpFilterHostToDevice(std::valarray<std::complex<T>> &host_filter)
{
    if (!_filter_set) {
        size_t sz_filter = host_filter.size()*sizeof(gpuComplex<T>);
        // allocate input
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&_d_filter), sz_filter));
        // copy input
        checkCudaErrors(hipMemcpy(_d_filter, &host_filter[0], sz_filter, hipMemcpyHostToDevice));
        _filter_set = true;
    }
}

template<class T>
void gpuFilter<T>::
writeFilter(size_t ncols, size_t nrows)
{
    std::valarray<std::complex<T>> filter;
    cpFilterHostToDevice(filter);
    isce::io::Raster filterRaster("filter.bin", ncols, nrows, 1, GDT_CFloat32, "ENVI");
    filterRaster.setBlock(filter, 0, 0, ncols, nrows);
}

template<class T>
__global__ void phaseShift_g(gpuComplex<T> *slc, 
        T *range, 
        double pxlSpace, 
        T conj, 
        double wavelength, 
        T wave_div, 
        int n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_elements) {
        T phase = 4.0*M_PI*pxlSpace*range[i]/wavelength;
        gpuComplex<T> complex_phase(cos(phase/wave_div), conj*sin(phase/wave_div));
        slc[i] *= complex_phase;
    }
}

template<>
__global__ void phaseShift_g<float>(gpuComplex<float> *slc, 
        float *range, 
        double pxlSpace, 
        float conj, 
        double wavelength, 
        float wave_div, 
        int n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_elements) {
        float phase = 4.0*M_PI*pxlSpace*range[i]/wavelength;
        gpuComplex<float> complex_phase(cosf(phase/wave_div), conj*sinf(phase/wave_div));
        slc[i] *= complex_phase;
    }
}

template<class T>
__global__ void filter_g(gpuComplex<T> *signal, gpuComplex<T> *filter, int n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_elements) {
        signal[i] *= filter[i];
    }
}

template<class T>
__global__ void sumSpectrum_g(gpuComplex<T> *spectrum, T *spectrum_sum, int n_rows, int n_cols)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_cols) {
        for (int i_row = 0; i_row < n_rows; ++i_row) {
            spectrum_sum[i] += abs(spectrum[i_row*n_cols + i]);
        }
    }
}

// DECLARATIONS
template class gpuFilter<float>;

template __global__ void
sumSpectrum_g<float>(gpuComplex<float> *spectrum, float *spectrum_sum, int n_rows, int n_cols);
